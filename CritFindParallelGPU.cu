
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>


#include<string>
#include<fstream>
#include<time.h>

using std::cout;
using std::string;
using std::ifstream;

#define min(a, b) (((a) < (b)) ? (a) : (b))

//Parametros da GPU
#define threadspblock 1024

//Parametros da Entrada
string in_file_name = "Caso.txt";

//Parametros da Saida
string out_file_name = "Saida.csv";
int sol_Size = 10000;

//Parametros do Sitema
int n_MU; // Numero de Unidades de medicao no plano
int n_meas; // Numero de medidas do plano
int kmax; //Cardinalidade maxima avaliada

//Estutura da matriz de resultados de combinações (Cn)
string Cn_file_name = "combs1000em5.txt";
const int n_rows_Cn=1001;
const int n_colums_Cn=6;
long long int Cn[(n_rows_Cn) * (n_colums_Cn)] = { 0 };

// Numero de combinacoes avaliadas simultaneamentes
const long long int wave_size = (int)pow(2, 20);
int card;
long long int n_combs_in_wave;

int max_mat_size=0;

//Variaveis de percentual
bool p25 = 0;
bool p50 = 0;
bool p75 = 0;

int y=0;

__device__ long long int *d_Cn;//Combinacoes pré calculadas
void load_case (double* &E, int* &meas_plan, int* &UMs);

void show_completition_percentage(long long int n_analysed_combs);

//Funcs da GPU
__global__ void step1_enumeration(int* combs, long long int combs_first_id,int n_MU, int card,long long int n_combs_in_wave, int kmax);


int main()
{ 
    int* UMs;     //Unidades de Medicao [n_MU]
    int* meas_plan; //Plano de medição [n_meas x 7]
    double* E;     // Matriz Covariancia E [n_meas x n_meas]
    load_case(E,meas_plan,UMs);
    cout<<"nMUs: " << n_MU << "; nMeds: " <<n_meas<< "; kmax: " <<kmax<< '\n';

    
    // Conjunto Solucao
    //int n_sols=0;
    int* Sols;
    Sols = (int*)malloc(kmax * sol_Size * sizeof(int));

    //Combinacoes de elementos
    int* combs; 
    combs = (int*)malloc((size_t)wave_size * kmax * sizeof(int));

    // Vetor booleano  1: Combinacao critica 0: Combinacao nao
    int* is_crit; 
    is_crit = (int*)malloc(wave_size * sizeof(int));
    for(int i = 0; i<wave_size;i++) is_crit[i] = 1;

    //Alocacoes na GPU
	double *d_E;// Matrix de Covariancia
	hipMalloc(&d_E,n_meas*n_meas * sizeof(double));
	hipMemcpy(d_E,E,n_meas*n_meas * sizeof(double),hipMemcpyHostToDevice);

    hipMalloc(&d_Cn,(n_rows_Cn)*(n_colums_Cn)*sizeof( long long int));
    hipMemcpyToSymbol(HIP_SYMBOL(d_Cn),&Cn,(n_rows_Cn)*(n_colums_Cn)* sizeof(long long int),0,hipMemcpyHostToDevice);
	int *d_combs;//Matriz com combinacoes enumeradas
    hipMalloc(&d_combs,wave_size*kmax * sizeof(int));
	
    int *d_isCrit;//Matris que indica combinacoes criticas
	hipMalloc(&d_isCrit,wave_size* sizeof(int));
	hipMemset( d_isCrit,1,wave_size* sizeof(int));

    int* d_UMs;     //Unidades de Medicao [n_MU]
    hipMalloc(&d_UMs,n_MU*sizeof(int));
    hipMemcpy(d_UMs,UMs,n_MU*sizeof(int),hipMemcpyHostToDevice);

    int* d_meas_plan; //Plano de medição [n_meas x 7]
    hipMalloc(&d_meas_plan,n_meas*7*sizeof(int));
    hipMemcpy(d_meas_plan,meas_plan,n_meas*7*sizeof(int),hipMemcpyHostToDevice);

    // Numero de combinacoes avaliadas simultaneamentes
    int card;
    long long int n_combs_in_wave;

    for (card = 1; card <=kmax ;card++)
    {
        cout << "->Cardinalidade " << card<<'\n';
        cout << "Iniciado...\n";
        
        long long int n_analysed_combs = 0; // Combinacoes vizitadas em todas as ondas
        //FILE* combs_file;
        while (n_analysed_combs < Cn[n_MU * (n_colums_Cn) + card])
        {
            n_combs_in_wave = min(wave_size, Cn[n_MU * (n_colums_Cn) + card] - n_analysed_combs);
            //Enumerar
            step1_enumeration<<<wave_size/threadspblock,threadspblock>>>(d_combs,n_analysed_combs+1,n_MU,card,n_combs_in_wave,kmax);
            hipMemcpy(combs,d_combs,wave_size*kmax * sizeof(int),hipMemcpyDeviceToHost);
            for (int i = 0; i<n_combs_in_wave; i++)
            {
                for (int j = 0; j < kmax; j++)
                {
                    cout<<combs[i*kmax + j];
                }
                cout<<'\n';
            }
            //Prop. 2
            //Prop. 1
            //Atualização

            n_analysed_combs += wave_size;
            //Printa percentuais para acompanhar andamento da analise de criticalidades
            show_completition_percentage(n_analysed_combs);
            
        }
        
        printf("\nFinalizado!\n"); p25=0; p50=0;p75=0; 
        //fclose(combs_file);
    }
    
    free(UMs);
    free(meas_plan);
    free(E);
    free(combs);
    free(is_crit);
    free(Sols);
   
}

void load_case(double* &E, int* &meas_plan, int* &UMs)
{
    // Combinacoes pre calculadas (Cn)
    ifstream Cnfile(Cn_file_name.c_str());
	for (int i = 0; i < n_rows_Cn; i++)
	{
		for (int j = 0; j < n_colums_Cn; j++)
		{
			Cnfile >> Cn[i * (n_colums_Cn) + j];
		}
	}
	Cnfile.close();

    //Entrada programa
    ifstream in_file(in_file_name);

    in_file >> n_MU;
    in_file >> n_meas;
    in_file >> kmax;
    
    //Lista de unidades de medicao avaliadas
    UMs = (int*)malloc(n_MU*sizeof(int));
    for (int j = 0; j < n_MU; j++)
    {
        in_file >> UMs[j];
    }
    
    //Leitura do plano de med
    meas_plan = (int*)malloc(n_meas * 7 *sizeof(int));
    for (int i = 0; i < n_meas; i++)
	{
		for (int j = 0; j < 7; j++)
		{
			in_file >> meas_plan[i * 7 + j];
		}
	}

    //Leitura da Matriz E
    E = (double*)malloc(n_meas * n_meas * sizeof(double));
    for (int i = 0; i < n_meas; i++)
	{
		for (int j = 0; j < n_meas; j++)
		{
			in_file >> E[i * n_meas + j];
		}
	}
}

void show_completition_percentage(long long int n_analysed_combs)
{
    if (n_analysed_combs > Cn[n_MU * (n_colums_Cn) + card]/4 && p25==0)
    {
        printf("25%% ->");
        p25 = 1;
    }
    if (n_analysed_combs > Cn[n_MU * (n_colums_Cn) + card]/2 && p50==0)
    {
        printf("50%% ->");
        p50 = 1;
    }
    if (n_analysed_combs > Cn[n_MU * (n_colums_Cn) + card] * 3 / 4 && p75==0)
    {
        printf("75%%");
        p75 = 1;
    }
}

__global__ void step1_enumeration(int* combs, long long int combs_first_id, int n_MU, int card, long long int n_combs_in_wave, int kmax)
{
    int linha = threadIdx.x + blockDim.x*blockIdx.x;
    printf("%lli ", d_Cn[5]);
    // if(linha<n_combs_in_wave)
    // {  
    //     int nZ = n_MU - card;
    //     int nO = card;
    //     long long int n = linha + combs_first_id;
    //     for (long long int i = 0; i < n_MU; i++)
    //     {
    //         nZ--;
    //         if(nZ>=0){
    //             long long int zcomb = d_Cn[(n_MU - 1 - i) * (n_colums_Cn) +  min(nO,nZ)];
    //             if (zcomb < n)
    //             {
    //                 combs[linha * kmax + (card-nO)] = i;
    //                 nO--;
    //                 nZ++;
    //                 n = n - zcomb;
    //             }
    //         }else
    //         {
    //             combs[linha * kmax + (card-nO)] = i;
    //             nO --;
    //         }
    //     }
    //     for(int j=card; j<kmax;j++) {
    //          combs[linha * kmax + j] = -1;
    //     }
    // }
    
}  