
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int d_c[3] = {3,2,1};
__global__ void hello_const(){
    printf("%i %i %i, ",d_c[0],d_c[1],d_c[2]);
}

int main(){
    int c[3] = {5,4,2};
    hipMemcpyToSymbol(HIP_SYMBOL(d_c),&c,3*sizeof(int),0,hipMemcpyHostToDevice);
    hello_const<<<2,4>>>();
}